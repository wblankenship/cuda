
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 3
#define N 3
#define P 3

__global__ void kernel(float*,float*,float*);
void random_floats(float*,int);
void print_matrix(float*,int,int);

int main(int argc,char** argv) {
  /**
   * Init all variables
   */
  int a_size      = sizeof(float)*M*N,
      b_size      = sizeof(float)*N*P,
      result_size = sizeof(float)*M*P;

  float a[]        = {1,2,3,4,5,6,7,8,9},
        b[]        = {9,8,7,6,5,4,3,2,1},
        answer[]   = {30,24,18,84,69,54,138,114,90},
        *result    = (float*)malloc(result_size),
        *d_a,
        *d_b,
        *d_result;

  /**
   * Setup device memory
   */
  hipMalloc((void**)&d_a,a_size);
  hipMalloc((void**)&d_b,b_size);
  hipMalloc((void**)&d_result,result_size);
  hipMemcpy(d_a,a,a_size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,b_size,hipMemcpyHostToDevice);

  /**
   * Start GPU
   */
  kernel<<<M,P>>>(d_a,d_b,d_result);

  /**
   * Copy results back to host
   */
  hipMemcpy(result,d_result,sizeof(float)* M * P,hipMemcpyDeviceToHost);

  /**
   * Print results
   */
  printf("Result: \n");
  print_matrix(result,M,P);
  printf("Expected: \n");
  print_matrix(answer,M,P);

  /**
   * Cleanup memory
   */
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  free(result);
  return 0;
}

void print_matrix(float *a,int cols,int rows) {
  int i,j;
  for(i=0;i<cols;i++) {
    for(j=0;j<rows;j++)
      printf("%f ",a[i*M+j]);
    printf("\n");
  }
}

__global__ void kernel(float *a,float *b,float *result) {
  int row = blockIdx.x,
      col = threadIdx.x,
      i;

  for(result[row*M+col]=0,i=0;i<N;i++) {
    result[row*M+col] += a[row*M+i] * b[i*N+col];
  }
}

void random_floats(float* a, int size) {
  int i;
  for(i=0;i<size;i++)
    a[i] = rand() % 8 + 1; //generate a number betwee 1 and 9
}
